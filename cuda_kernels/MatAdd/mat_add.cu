// 2D addition

#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void MatAdd(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row * N + col < N * N)
        C[row * N + col] = A[row * N + col] + B[row * N + col];
}

int main() {
    float *A, *B, *C;

    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * N * sizeof(float));
    C = (float*)malloc(N * N * sizeof(float));

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = i;
            B[i * N + j] = j;
        }
    }

    float *refC;
    refC = (float*)malloc(N * N * sizeof(float));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            refC[i * N + j] = A[i * N + j] + B[i * N + j];
        }
    }
    
    // Device-side allocation for d_A, d_B, d_C
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    bool mismatch = false;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (C[i * N + j] != refC[i * N + j]) {
                std::cout << "Mismatch at (" << i << ", " << j << ")" << std::endl;
                mismatch = true;
                break;
            }
        }
        if (mismatch) {
            break;
        }
    }

    if (mismatch) {
        std::cout << "Mismatch found!" << std::endl;
    } else {
        std::cout << "No mismatch found!" << std::endl;
    }
    
    // Free the allocated memory
    free(A);
    free(B);
    free(C);
    free(refC);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}
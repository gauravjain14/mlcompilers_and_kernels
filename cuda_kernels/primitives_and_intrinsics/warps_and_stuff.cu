
#include <hip/hip_runtime.h>
#include <stdio.h>


// __shfl_sync, __shfl_up_sync, __shfl_down_sync, __shfl_xor_sync exchange
// a variable between threads within a warp
__global__ void bcast(int arg, int *fill_values) {
    int laneId = threadIdx.x & 0x1f;
    int value;
    if (laneId == 0)        // Note unused variable for
        value = arg;        // all threads except lane 0
    value = __shfl_sync(0xffffffff, value, 0);
    fill_values[threadIdx.x] = value;
}


// __shfl_up_sync -- calculates source lane id by subtracting delta from
// the caller's lane id.
// one use case of this can be that lane 0 values are exchanged with lane 4,
// lane 1 values are exchanged with lane 5 --> This essentially can move values up.
__global__ void exchange(int *fill_values, int delta) {
    int laneId = threadIdx.x & 0x1f;
    int value = fill_values[threadIdx.x];
    value = __shfl_up_sync(0xffffffff, value, delta);
    fill_values[threadIdx.x] = value;
}


__global__ void butterfly(int* input_data, int* output_sums, int num_elements, long long *active_mask) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= num_elements) {
        return;
    }

    int laneId = threadIdx.x & 0x1f;
    int calling_value = input_data[tid];

    // Get the active threads mask here before divergence.
    // Could an alternative here be to use __ballot_sync?
    unsigned int active_threads_mask = __activemask();
    // Getting the mask here also gives the right value in active_mask!
    active_mask[0] = active_threads_mask;

    for (int i = 1; i < 32; i <<= 1) {
        // Use the mask 0xFFFFFFFF to ensure that all threads participate in the computation.
        int partner_value = __shfl_xor_sync(active_threads_mask, calling_value, i, 32);
        calling_value += partner_value;
    }

    // After the loop, only lane 0 of each warp will hold the total sum
    // of all values that were originally in that warp.
    if (laneId == 0) {
        output_sums[blockIdx.x] = calling_value;
    }
}


int main() {
    int h_values[32];
    for (int i = 0; i < 32; i++) {
        h_values[i] = i;
    }

    int *d_values;
    hipMalloc(&d_values, 32 * sizeof(int));
    hipMemcpy(d_values, h_values, 32 * sizeof(int), hipMemcpyHostToDevice);

    bcast<<< 1, 32 >>>(1234, d_values);
    hipDeviceSynchronize();

    hipMemcpy(h_values, d_values, 32 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 32; i++) {
        printf("%d ", h_values[i]);
    }   
    printf("\n");

    for (int i = 0; i < 32; i++) {
        h_values[i] = i;
    }

    hipMemcpy(d_values, h_values, 32 * sizeof(int), hipMemcpyHostToDevice);
    exchange<<< 1, 32 >>>(d_values, 4);
    hipDeviceSynchronize();
    
    hipMemcpy(h_values, d_values, 32 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 32; i++) {
        printf("%d ", h_values[i]);
    }
    printf("\n");

    for (int i = 0; i < 32; i++) {
        h_values[i] = i;
    }

    // Allocate output array for butterfly results
    int *d_output_sums;
    hipMalloc(&d_output_sums, sizeof(int));  // Only need 1 int for the sum
    
    hipMemcpy(d_values, h_values, 32 * sizeof(int), hipMemcpyHostToDevice);
    long long *d_active_mask;
    hipMalloc(&d_active_mask, sizeof(long long));
    butterfly<<< 1, 32 >>>(d_values, d_output_sums, 32, d_active_mask);
    hipDeviceSynchronize();
    
    // Copy back the sum result
    int h_sum;
    hipMemcpy(&h_sum, d_output_sums, sizeof(int), hipMemcpyDeviceToHost);
    
    // Calculate expected sum on CPU for verification
    int expected_sum = 0;
    for (int i = 0; i < 32; i++) {
        expected_sum += i;  // Sum of 0+1+2+...+31
    }
    
    printf("Butterfly GPU result: %d\n", h_sum);
    printf("Expected CPU result: %d\n", expected_sum);
    printf("Verification: %s\n", (h_sum == expected_sum) ? "PASS" : "FAIL");
    printf("\n");

    // Test the butterfly reduction with 6 elements
    int h_values_6[12];
    for (int i = 0; i < 12; i++) {
        h_values_6[i] = i;
    }
    int *d_values_6;
    hipMalloc(&d_values_6, 12 * sizeof(int));
    hipMemcpy(d_values_6, h_values_6, 12 * sizeof(int), hipMemcpyHostToDevice);

    long long *d_active_mask_6;
    hipMalloc(&d_active_mask_6, sizeof(long long));

    int *d_output_sums_6;
    hipMalloc(&d_output_sums_6, sizeof(int));
    butterfly<<< 1, 12 >>>(d_values_6, d_output_sums_6, 12, d_active_mask_6);
    hipDeviceSynchronize();

    int h_sum_6;
    hipMemcpy(&h_sum_6, d_output_sums_6, sizeof(int), hipMemcpyDeviceToHost);

    long long h_active_mask;
    hipMemcpy(&h_active_mask, d_active_mask_6, sizeof(long long), hipMemcpyDeviceToHost);

    int expected_sum_6 = 0;
    for (int i = 0; i < 12; i++) {
        expected_sum_6 += i;
    }

    printf("Butterfly GPU result: %d\n", h_sum_6);
    printf("Expected CPU result: %d\n", expected_sum_6);
    printf("Verification: %s\n", (h_sum_6 == expected_sum_6) ? "PASS" : "FAIL");
    printf("Active mask: %lld\n", h_active_mask);
    printf("\n");

    hipFree(d_values);
    hipFree(d_output_sums);
    hipFree(d_values_6);
    return 0;
}